//  Create on: 2016/10/24 ShanghaiTech
//  Author:    Yingying Zhang

#include <vector>

#include "caffe/layers/scale_invariant_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void ScaleInvariantLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  // 1. sum(d_i ^ 2) / 2n
  int count = bottom[0]->count();
  caffe_gpu_sub(count, bottom[0]->gpu_data(), bottom[1]->gpu_data(),
                diff_.mutable_gpu_data());
  Dtype dot;
  caffe_gpu_dot(count, diff_.gpu_data(), diff_.gpu_data(), &dot);
  Dtype loss = dot / bottom[0]->num() / Dtype(2);
  // 2. lambda  * (sum(d_i) ^ 2 / 2n^2
  caffe_gpu_set(count, Dtype(1), ones_.mutable_gpu_data());
  caffe_gpu_dot(count, diff_.gpu_data(), ones_.gpu_data(), &sum_di_);
  loss -= lambda_ * sum_di_ * sum_di_ / bottom[0]->num() / bottom[0]->num() /
          Dtype(2);
  top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
void ScaleInvariantLossLayer<Dtype>::Backward_gpu(
    const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  for (int i = 0; i < 2; ++i) {
    if (propagate_down[i]) {
      // 1. gradient for sum(d_i ^ 2) / 2n
      const Dtype sign = (i == 0) ? 1 : -1;
      Dtype alpha = sign * top[0]->cpu_diff()[0] / bottom[i]->num();
      caffe_gpu_axpby(bottom[i]->count(), alpha, diff_.gpu_data(), Dtype(0),
                      bottom[i]->mutable_gpu_diff());
      // 2. graidient for lambda  * (sum(d_i) ^ 2 / 2n^2
      alpha = -sign * top[0]->cpu_diff()[0] * sum_di_ * lambda_ /
              bottom[i]->num() / bottom[i]->num();
      caffe_gpu_axpby(bottom[i]->count(), alpha, ones_.gpu_data(),
                      Dtype(1),  // accumulate gradient
                      bottom[i]->mutable_gpu_diff());
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(ScaleInvariantLossLayer);

}  // namespace caffe
