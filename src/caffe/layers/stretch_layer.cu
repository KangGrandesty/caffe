#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/stretch_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void StretchForward(const int n, const Dtype *in, Dtype *out,
                               const int channels_total, const int width) {
  CUDA_KERNEL_LOOP(index, n) {
    int offset = (index / width) % 2;
    int count_channels = index / channels_total;
    out[(count_channels * 2 + (index + offset) % 2) * channels_total +
        index % channels_total] = in[index];
  }
}

template <typename Dtype>
void StretchLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype> *> &bottom,
                                      const vector<Blob<Dtype> *> &top) {
  const Dtype *in_data = bottom[0]->gpu_data();
  const int count_ = bottom[0]->count();
  Dtype *out_data = top[0]->mutable_gpu_data();
  const vector<int> shape = bottom[0]->shape();
  const int channels_total = shape[2] * shape[3];
  caffe_gpu_set(count_ * 2, Dtype(0), out_data);
  StretchForward<Dtype><<<CAFFE_GET_BLOCKS(count_), CAFFE_CUDA_NUM_THREADS>>>(
      count_, in_data, out_data, channels_total, shape[3]);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void StretchBackward(const int n, const Dtype *in, Dtype *out,
                                const int channels_total, const int width) {
  CUDA_KERNEL_LOOP(index, n) {
    int offset = (index / width) % 2;
    int count_channels = index / channels_total;
    out[index] =
        in[(count_channels * 2 + (index + offset) % 2) * channels_total +
               index % channels_total];
  }
}

template <typename Dtype>
void StretchLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype> *> &top,
                                       const vector<bool> &propagate_down,
                                       const vector<Blob<Dtype> *> &bottom) {
  if (propagate_down[0]) {
    const Dtype *in_diff = top[0]->gpu_diff();
    const int count_ = bottom[0]->count();
    Dtype *out_diff = bottom[0]->mutable_gpu_diff();
    const vector<int> shape = bottom[0]->shape();
    const int channels_total = shape[2] * shape[3];
    StretchBackward<
        Dtype><<<CAFFE_GET_BLOCKS(count_), CAFFE_CUDA_NUM_THREADS>>>(
        count_, in_diff, out_diff, channels_total, shape[3]);
    CUDA_POST_KERNEL_CHECK;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(StretchLayer);

}  // namespace caffe
