#include "hip/hip_runtime.h"
//
// Created by admins on 17-10-8.
//
#include <vector>

#include "caffe/layers/flat_patch_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void FlatPatchForeWard(const int n, const Dtype *in, Dtype *out,
                                  const int num_total, const int channels_total,
                                  const int width, const int size_h,
                                  const int size_w) {
  const int channels_total_ = channels_total / size_w / size_h;
  const int width_out = width / size_w;
  CUDA_KERNEL_LOOP(index, n) {
    int count_num = index / num_total;
    int count_channels = (index % num_total) / channels_total;
    int height_data = (index % channels_total) / width;
    int width_data = index % width;
    int seat_height = height_data / size_h;
    int count_height = height_data % size_h;
    int seat_width = width_data / size_w;
    int count_width = width_data % size_w;
    out[count_num * num_total +
        ((count_channels * size_h + count_height) * size_w + count_width) *
            channels_total_ +
        seat_height * width_out + seat_width] = in[index];
  }
}

template <typename Dtype>
void FlatPatchLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype> *> &bottom,
                                        const vector<Blob<Dtype> *> &top) {
  const vector<int> shape = bottom[0]->shape();
  const int count_ = bottom[0]->count();
  const Dtype *in_data = bottom[0]->gpu_data();
  Dtype *out_data = top[0]->mutable_gpu_data();
  FlatPatchForeWard<
      Dtype><<<CAFFE_GET_BLOCKS(count_), CAFFE_CUDA_NUM_THREADS>>>(
      count_, in_data, out_data, shape[1] * shape[2] * shape[3],
      shape[2] * shape[3], shape[3], size_h, size_w);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void FlatPatchBackWard(const int n, const Dtype *in, Dtype *out,
                                  const int num_total, const int channels_total,
                                  const int width, const int size_h,
                                  const int size_w) {
  const int channels_total_ = channels_total / size_w / size_h;
  const int width_out = width / size_w;
  CUDA_KERNEL_LOOP(index, n) {
    int count_num = index / num_total;
    int count_channels = (index % num_total) / channels_total;
    int height_data = (index % channels_total) / width;
    int width_data = index % width;
    int seat_height = height_data / size_h;
    int count_height = height_data % size_h;
    int seat_width = width_data / size_w;
    int count_width = width_data % size_w;
    out[index] =
        in[count_num * num_total +
           ((count_channels * size_h + count_height) * size_w + count_width) *
               channels_total_ +
           seat_height * width_out + seat_width];
  }
}

template <typename Dtype>
void FlatPatchLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype> *> &top,
                                         const vector<bool> &propagate_down,
                                         const vector<Blob<Dtype> *> &bottom) {
  if (propagate_down[0]) {
    const vector<int> shape = bottom[0]->shape();
    const int count_ = bottom[0]->count();
    const Dtype *in_diff = top[0]->gpu_diff();
    Dtype *out_diff = bottom[0]->mutable_gpu_diff();
    FlatPatchBackWard<
        Dtype><<<CAFFE_GET_BLOCKS(count_), CAFFE_CUDA_NUM_THREADS>>>(
        count_, in_diff, out_diff, shape[1] * shape[2] * shape[3],
        shape[2] * shape[3], shape[3], size_h, size_w);
    CUDA_POST_KERNEL_CHECK;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(FlatPatchLayer);

}  // namespace caffe
