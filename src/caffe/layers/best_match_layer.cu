#include "hip/hip_runtime.h"
#include <numeric>
#include <vector>

#include "caffe/layers/best_match_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void BestMatchForward(const int n, const Dtype *in1,
                                 const Dtype *in2, const Dtype *match,
                                 Dtype *out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = ((in1[index] + in2[index] - Dtype(2) * match[index]) *
                  (in1[index] - in2[index])) > Dtype(0)
                     ? in2[index]
                     : in1[index];
  }
}

template <typename Dtype>
void BestMatchLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype> *> &bottom,
                                        const vector<Blob<Dtype> *> &top) {
  const Dtype *cand_data1 = bottom[0]->gpu_data();
  const Dtype *cand_data2 = bottom[1]->gpu_data();
  const Dtype *match_data = bottom[2]->gpu_data();
  const int count = bottom[0]->count();
  Dtype *top_data = top[0]->mutable_gpu_data();
  if (this->phase_ == TRAIN) {
    BestMatchForward<
        Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, cand_data1, cand_data2, match_data, top_data);
    CUDA_POST_KERNEL_CHECK;
  } else {
    caffe_copy(count, cand_data1, top_data);
    caffe_gpu_axpy(count, Dtype(1), cand_data2, top_data);
  }
}

template <typename Dtype>
__global__ void BestMatchBackward(const int n, const Dtype *in1,
                                  const Dtype *in2, const Dtype *data,
                                  const Dtype *diff, Dtype *out1, Dtype *out2) {
  CUDA_KERNEL_LOOP(index, n) {
    out1[index] = (in1[index] == data[index]) * diff[index];
    out2[index] = (in2[index] == data[index]) * diff[index];
  }
}

template <typename Dtype>
void BestMatchLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype> *> &top,
                                         const vector<bool> &propagate_down,
                                         const vector<Blob<Dtype> *> &bottom) {
  const Dtype *cand_data1 = bottom[0]->gpu_data();
  const Dtype *cand_data2 = bottom[1]->gpu_data();
  const Dtype *top_diff = top[0]->gpu_diff();
  const Dtype *top_data = top[0]->gpu_data();
  const int count = bottom[0]->count();
  Dtype *cand_diff1 = bottom[0]->mutable_gpu_diff();
  Dtype *cand_diff2 = bottom[1]->mutable_gpu_diff();
  Dtype *match_diff2 = bottom[2]->mutable_gpu_diff();
  caffe_gpu_set(count, Dtype(0), match_diff2);
  BestMatchBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, cand_data1, cand_data2, top_data, top_diff, cand_diff1,
      cand_diff2);
  CUDA_POST_KERNEL_CHECK;
}

INSTANTIATE_LAYER_GPU_FUNCS(BestMatchLayer);

}  // namespace caffe
