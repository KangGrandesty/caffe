#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/relnu_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void ReLnUForward(const int n, const Dtype* in, Dtype* out,
    Dtype alpha, Dtype beta)
{
    CUDA_KERNEL_LOOP(index, n)
    {
        out[index] = in[index] > Dtype(0) ? (beta * std::log(alpha * in[index] + 1)) : Dtype(0);
    }
}

template <typename Dtype>
void ReLnULayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top)
{
    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();
    const int count = bottom[0]->count();
    Dtype alpha = this->layer_param_.relnu_param().alpha();
    Dtype beta = this->layer_param_.relnu_param().beta();
    ReLnUForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, top_data, alpha, beta);
    CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void ReLnUBackward(const int n, const Dtype* in_diff,
    const Dtype* in_data, Dtype* out_diff, Dtype alpha, Dtype beta)
{
    CUDA_KERNEL_LOOP(index, n)
    {
        out_diff[index] = in_diff[index] * (in_data[index] > Dtype(0) ? (beta / (in_data[index] + Dtype(1) / alpha)) : Dtype(0));
    }
}

template <typename Dtype>
void ReLnULayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom)
{
    if (propagate_down[0]) {
        const Dtype* bottom_data = bottom[0]->gpu_data();
        const Dtype* top_diff = top[0]->gpu_diff();
        Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
        const int count = bottom[0]->count();
        Dtype alpha = this->layer_param_.relnu_param().alpha();
        Dtype beta = this->layer_param_.relnu_param().beta();
        // NOLINT_NEXT_LINE(whitespace/operators)
        ReLnUBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
            count, top_diff, bottom_data, bottom_diff, alpha, beta);
        CUDA_POST_KERNEL_CHECK;
    }
}

INSTANTIATE_LAYER_GPU_FUNCS(ReLnULayer);

} // namespace caffe
