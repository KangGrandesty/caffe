#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/shift_relu_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SReLUShift(Dtype* shifts, const Dtype* data, const int n,
                           const int offset) {
  CUDA_KERNEL_LOOP(index, n) {
    shifts[index / offset] +=
        data[index] > Dtype(0) ? data[index] : -data[index];
  }
}

template <typename Dtype>
__global__ void SReLUForward(const int n, const int offset, const Dtype* in,
                             Dtype* out, const Dtype negative_slope,
                             const Dtype* shifts) {
  CUDA_KERNEL_LOOP(index, n) {
    const Dtype shift = shifts[index / offset];
    const Dtype data = in[index] - shift;
    out[index] = data > Dtype(0) ? data : (data * negative_slope);
  }
}

template <typename Dtype>
void SReLULayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const Dtype negative_slope =
      this->layer_param_.shift_relu_param().negative_slope();
  const Dtype shift = this->layer_param_.shift_relu_param().shift();
  const Dtype range = this->layer_param_.shift_relu_param().range();
  const int count = bottom[0]->count();
  if (this->phase_ == TRAIN && range > Dtype(0)) {
    SReLUShift<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        mask_.mutable_gpu_data(), bottom_data, count, offset);
    caffe_gpu_scal(shift_.count(), Dtype(1) / offset, mask_.mutable_gpu_data());
    caffe_gpu_rng_uniform(shift_.count(), -range, range,
                          bias_.mutable_gpu_data());
    caffe_gpu_mul(shift_.count(), bias_.gpu_data(), mask_.gpu_data(),
                  shift_.mutable_gpu_data());
    caffe_gpu_add_scalar(shift_.count(), shift, shift_.mutable_gpu_data());
  } else {
    caffe_gpu_set(shift_.count(), shift, shift_.mutable_gpu_data());
  }
  SReLUForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, offset, bottom_data, top_data, negative_slope, shift_.gpu_data());
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void SReLUBackward(const int n, const int offset,
                              const Dtype* in_diff, const Dtype* in_data,
                              Dtype* out_diff, const Dtype negative_slope,
                              const Dtype* shifts) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] =
        in_diff[index] *
        (in_data[index] > shifts[index / offset] ? Dtype(1) : negative_slope);
  }
}

template <typename Dtype>
void SReLULayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                                     const vector<bool>& propagate_down,
                                     const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const Dtype negative_slope =
        this->layer_param_.shift_relu_param().negative_slope();
    const int count = bottom[0]->count();
    const Dtype* shifts = shift_.mutable_gpu_data();
    SReLUBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, offset, top_diff, bottom_data, bottom_diff, negative_slope,
        shifts);
    CUDA_POST_KERNEL_CHECK;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SReLULayer);

}  // namespace caffe
