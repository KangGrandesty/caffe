#include "hip/hip_runtime.h"

#include <vector>

#include "caffe/layers/polarize_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void PolarizeForward(const int n, const Dtype* in,
                                const unsigned int* mask,
                                const unsigned int threshold,
                                const Dtype zoom_up_, const Dtype zoom_down_,
                                Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = mask[index] > threshold ? (in[index] * zoom_up_)
                                         : (in[index] * zoom_down_);
    //    out[index] = in[index] * zoom[mask[index] > threshold];
  }
}

template <typename Dtype>
void PolarizeLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                       const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  if (this->phase_ == TRAIN) {
    unsigned int* mask =
        static_cast<unsigned int*>(rand_vec_.mutable_gpu_data());
    caffe_gpu_rng_uniform(count, mask);
    //    const Dtype zoom[2] = {zoom_[0], zoom_[1]};
    // set thresholds
    // NOLINT_NEXT_LINE(whitespace/operators)
    PolarizeForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, mask, uint_thres_, zoom_[1], zoom_[0], top_data);
    CUDA_POST_KERNEL_CHECK;
  } else {
    caffe_copy(count, bottom_data, top_data);
  }
}

template <typename Dtype>
__global__ void PolarizeBackward(const int n, const Dtype* in_diff,
                                 const unsigned int* mask,
                                 const unsigned int threshold,
                                 const Dtype zoom_up_, const Dtype zoom_down_,
                                 Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = mask[index] > threshold ? (in_diff[index] * zoom_up_)
                                              : (in_diff[index] * zoom_down_);
    //    out_diff[index] = in_diff[index] * zoom[mask[index] > threshold];
  }
}

template <typename Dtype>
void PolarizeLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                                        const vector<bool>& propagate_down,
                                        const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const unsigned int* mask =
        static_cast<const unsigned int*>(rand_vec_.gpu_data());
    const int count = bottom[0]->count();
    //    const Dtype zoom[2] = {zoom_[0], zoom_[1]};
    // NOLINT_NEXT_LINE(whitespace/operators)
    PolarizeBackward<
        Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, mask, uint_thres_, zoom_[1], zoom_[0], bottom_diff);
    CUDA_POST_KERNEL_CHECK;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(PolarizeLayer);

}  // namespace caffe
