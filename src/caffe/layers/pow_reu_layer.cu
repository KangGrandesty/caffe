#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/pow_reu_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void PowReUForward(const int n, const Dtype* in,
    Dtype* out, Dtype powers) {
    CUDA_KERNEL_LOOP(index, n) {
        out[index] = in[index] > Dtype(0) ? (std::pow(in[index] + Dtype(1), powers) - Dtype(1)) : Dtype(0);
    }
}

template <typename Dtype>
void PowReULayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top)
{
    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();
    const int count = bottom[0]->count();
    Dtype powers = this->layer_param_.pow_reu_param().powers();
    PowReUForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, top_data, powers);
    CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void PowReUBackward(const int n, const Dtype* in_diff, const Dtype* in_data,
    Dtype* out_diff, const Dtype* out_data, Dtype powers) {
    CUDA_KERNEL_LOOP(index, n) {
        out_diff[index] = in_diff[index] * (in_data[index] > Dtype(0) ? (powers * (out_data[index] + 1) / (in_data[index] + 1)) : Dtype(0));
    }
}

template <typename Dtype>
void PowReULayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom)
{
    if (propagate_down[0]) {
        const Dtype* bottom_data = bottom[0]->gpu_data();
        const Dtype* top_data = top[0]->gpu_data();
        const Dtype* top_diff = top[0]->gpu_diff();
        Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
        const int count = bottom[0]->count();
        Dtype powers = this->layer_param_.pow_reu_param().powers();
        // NOLINT_NEXT_LINE(whitespace/operators)
        PowReUBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
            count, top_diff, bottom_data, bottom_diff, top_data, powers);
        CUDA_POST_KERNEL_CHECK;
    }
}

INSTANTIATE_LAYER_GPU_FUNCS(PowReULayer);

} // namespace caffe
